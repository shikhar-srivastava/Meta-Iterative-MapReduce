#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "mpi.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define MCW MPI_COMM_WORLD

__global__ void someFunction(int *x) {

}

int main(int argc, char* argv[]) {
	MPI_Init(&argc, &argv);
	int rank, size;
	MPI_Comm_rank(MCW, &rank);
	MPI_Comm_size(MCW, &size);
	if (rank == 0) {

	}

	//scatter data
	int* x;
	hipMalloc((void**)&x, 10);
	someFunction<<<1,1>>>(x);
}